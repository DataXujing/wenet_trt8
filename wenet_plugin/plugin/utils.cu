#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipblas.h"
#include <string>
#include <vector>
#include <cassert>

template<typename T>
__global__ void transpose_4d_batch_major_mem_q_cache(
    T* v_dst, const T* v_src, const int batch_size, const int seq_length, const int d_model)
{
    // B, L,Dm -> L, B, Dm
    const int batch_id = blockIdx.y;
    const int seq_id = blockIdx.z;

    // 16 byte loads will handle "x" dimension
    auto val_src = reinterpret_cast<const uint4*>(v_src + batch_id * d_model * seq_length
                                                  + seq_id * d_model);
    auto val_dst = reinterpret_cast<uint4*>(v_dst + seq_id * batch_size * d_model
                                            + batch_id * d_model);

    // idx is over output dimension L * size_per_head / x for values
    const int out_idx = blockIdx.x * blockDim.x + threadIdx.x;

    constexpr int X_ELEMS = (sizeof(T) == 4) ? 4 : 8;
    const int size_limit = d_model / X_ELEMS;
    if (out_idx >= size_limit) {
        return;
    }

    val_dst[out_idx] = val_src[out_idx];
}

template<typename T>
void transpose_3d_102_memory_kernelLauncher(T* dst,
                                            const T* src,
                                            const int local_batch_size,
                                            const int max_seq_len,
                                            const int d_model,
                                            hipStream_t stream)
{
    constexpr int block_sz = 128;

    constexpr int x = (sizeof(T) == 4) ? 4 : 8;
    int size = d_model / x;
    dim3 grid((size + block_sz - 1) / block_sz, local_batch_size, max_seq_len);

    transpose_4d_batch_major_mem_q_cache<<<grid, block_sz, 0, stream>>>(
            dst, src, local_batch_size, max_seq_len, d_model);
}

template<typename T>
inline __device__ T ldg(const T* val) {
    return __ldg(val);
}

template<typename T>
__global__ void add_fusedQKV_bias_transpose_kernel(T* qkv_buf,
                                                   const T* __restrict QKV,
                                                   const T* __restrict qkv_bias,
                                                   const int batch_size,
                                                   const int seq_len,
                                                   const int head_num,
                                                   const int size_per_head)
{
    // QKV: [m, n]
    // qkv_bias: [n]
    // q_buf, k_buf, v_buf: [batch, head_num, seq_len, size_per_head]

    const int n = head_num * size_per_head;
    for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < batch_size * seq_len * n;
         index += gridDim.x * blockDim.x) {
        int bias_id = index % (n);
        T val = ldg(&QKV[index]) + ldg(&qkv_bias[bias_id]);

        int tmp_index = index;
        const int target_batch_id = tmp_index / (seq_len * n);
        tmp_index -= target_batch_id * seq_len * n;     //  current batch index
        const int seq_id = tmp_index / (n);             //  current seq id
        tmp_index -= seq_id * n;                        //  current batch, seq, nindex
        const int head_id = tmp_index / size_per_head;
        const int size_id = tmp_index - head_id * size_per_head;

        qkv_buf[target_batch_id * head_num * seq_len * size_per_head 
                + head_id * seq_len * size_per_head
                + seq_id * size_per_head + size_id] = val;
    }
}

template __global__ void add_fusedQKV_bias_transpose_kernel(float* qkv_buf,
                                            const float* __restrict QKV,
                                            const float* __restrict qkv_bias,
                                            const int batch_size,
                                            const int seq_len,
                                            const int head_num,
                                            const int size_per_head);


template<typename T>
__global__ void add_bias_kernel(T* qkv_buf,
                                const T* __restrict QKV,
                                const T* __restrict qkv_bias,
                                const int batch_size,
                                const int seq_len,
                                const int n)
{
    // QKV: [m, n]
    // qkv_bias: [n]

    for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < batch_size * seq_len * n;
         index += gridDim.x * blockDim.x) {
        int bias_id = index % (n);
        T val = ldg(&QKV[index]) + ldg(&qkv_bias[bias_id]);
        qkv_buf[index] = val;
    }
}

template<typename T>
__global__ void transpose_4d_batch_major_k_cache(
    T* k_dst, const T* k_src, const int head_num, const int size_per_head, const int seq_len, const int max_seq_len)
{
    const int batch_id = blockIdx.y;
    const int head_id = blockIdx.z;
    constexpr int X_ELEMS = (sizeof(T) == 4) ? 4 : 8;

    auto key_src = reinterpret_cast<const uint4*>(k_src + batch_id * head_num * size_per_head * seq_len
                                                  + head_id * size_per_head * seq_len);
    auto key_dst = reinterpret_cast<uint4*>(k_dst + batch_id * head_num * size_per_head * max_seq_len
                                            + head_id * size_per_head * max_seq_len);

    const int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size_per_head_div_x = size_per_head / X_ELEMS;
    if (out_idx >= size_per_head_div_x * max_seq_len) {
        return;
    }

    int idx = out_idx;
    const int k_seq_len_id = idx % max_seq_len;
    idx = (idx - k_seq_len_id) / max_seq_len;
    const int k_head_size_id = idx % size_per_head_div_x;

    if (k_seq_len_id < seq_len) {
        key_dst[out_idx] = key_src[k_seq_len_id * size_per_head_div_x + k_head_size_id];
    }
}

template<typename T>
__global__ void transpose_4d_batch_major_v_cache(
    T* v_dst, const T* v_src, const int head_num, const int size_per_head, const int seq_len, const int max_seq_len)
{
    const int batch_id = blockIdx.y;
    const int head_id = blockIdx.z;

    // 16 byte loads will handle "x" dimension
    auto val_src = reinterpret_cast<const uint4*>(v_src + batch_id * head_num * size_per_head * seq_len
                                                  + head_id * size_per_head * seq_len);
    auto val_dst = reinterpret_cast<uint4*>(v_dst + batch_id * head_num * size_per_head * max_seq_len
                                            + head_id * size_per_head * max_seq_len);

    // idx is over output dimension L * size_per_head / x for values
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    constexpr int X_ELEMS = (sizeof(T) == 4) ? 4 : 8;
    const int size_per_head_div_x = size_per_head / X_ELEMS;

    if (idx >= size_per_head_div_x * seq_len) {
        return;
    }

    val_dst[idx] = val_src[idx];
}

template<typename T>
void invokeTranspose4dBatchMajor(T* k_dst,
                                 T* v_dst,
                                 const T* k_src,
                                 const T* v_src,
                                 const int local_batch_size,
                                 const int seq_len,
                                 const int max_seq_len,
                                 const int size_per_head,
                                 const int local_head_num,
                                 hipStream_t stream)
{
    constexpr int block_sz = 128;
    constexpr int x = (sizeof(T) == 4) ? 4 : 8;
    int size = max_seq_len * size_per_head / x;
    dim3 grid((size + block_sz - 1) / block_sz, local_batch_size, local_head_num);
    dim3 grid_v((seq_len * size_per_head / x + block_sz - 1) / block_sz, local_batch_size, local_head_num);

    transpose_4d_batch_major_k_cache<<<grid, block_sz, 0, stream>>>(
        k_dst, k_src, local_head_num, size_per_head, seq_len, max_seq_len);

    transpose_4d_batch_major_v_cache<<<grid_v, block_sz, 0, stream>>>(
        v_dst, v_src, local_head_num, size_per_head, seq_len, max_seq_len);
}

static const float HALF_FLT_MAX = 65504.F;
#define FINAL_MASK 0xffffffff

template<typename T>
__inline__ __device__ T warpReduceSum(T val)
{
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1)
        val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
    return val;
}

/* Calculate the sum of all elements in a block */
template<typename T>
__inline__ __device__ T blockReduceSum(T val)
{
    static __shared__ T shared[32];
    int lane = threadIdx.x & 0x1f;
    int wid = threadIdx.x >> 5;

    val = warpReduceSum<T>(val);

    if (lane == 0)
        shared[wid] = val;

    __syncthreads();

    // Modify from blockDim.x << 5 to blockDim.x / 32. to prevent
    // blockDim.x is not divided by 32
    val = (threadIdx.x < (blockDim.x / 32.f)) ? shared[lane] : (T)(0.0f);
    val = warpReduceSum<T>(val);

    return val;
}

template<typename T>
__inline__ __device__ T warpReduceMax(T val)
{
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1)
        val = max(val, __shfl_xor_sync(FINAL_MASK, val, mask, 32));
    return val;
}

/* Calculate the maximum of all elements in a block */
template<typename T>
__inline__ __device__ T blockReduceMax(T val)
{
    static __shared__ T shared[32];
    int lane = threadIdx.x & 0x1f;  // in-warp idx
    int wid = threadIdx.x >> 5;     // warp idx

    val = warpReduceMax(val);  // get maxx in each warp

    if (lane == 0)  // record in-warp maxx by warp Idx
        shared[wid] = val;

    __syncthreads();

    // Modify from blockDim.x << 5 to blockDim.x / 32. to prevent
    // blockDim.x is not divided by 32
    val = (threadIdx.x < (blockDim.x / 32.f)) ? shared[lane] : -1e20f;
    val = warpReduceMax(val);

    return val;
}
// TODO(bhsueh) Rename the softmax_kernel_v4 to softmax_kernel
template<int ITEMS_PER_THREAD, typename T, typename T_IN>
__global__ void softmax_kernel_v4(T* qk_buf_,
                                  const T_IN* qk_buf_src,
                                  const int* attr_mask,
                                  const bool isCrossAtten,
                                  const int batch_size,
                                  const int head_num,
                                  const int seq_len,
                                  const int kv_len,
                                  const T scalar)
{
    for (int seq_id = blockIdx.x; seq_id < seq_len; seq_id += gridDim.x) {
        float data[ITEMS_PER_THREAD];
        int qk_offset;
        __shared__ float s_mean, s_max;
        float local_max = -1e20f;
        int mask_offset = blockIdx.y;
        if (isCrossAtten)
            mask_offset = blockIdx.y/10;
        int enc_len = static_cast<int>(ldg(&attr_mask[mask_offset]));
        if(!isCrossAtten){
            enc_len = enc_len>seq_id ? seq_id+1 : enc_len;
        }
        for (int i = 0; blockDim.x * i + threadIdx.x < kv_len; i++) {
            qk_offset =
                ((blockIdx.y * head_num + blockIdx.z) * seq_len + seq_id) * kv_len + blockDim.x * i + threadIdx.x;

            float qk = static_cast<float>(qk_buf_src[qk_offset]);


            float mask_val = (blockDim.x * i + threadIdx.x)<enc_len? 0.f : -10000.0f;

            data[i] = qk * static_cast<float>(scalar) + mask_val;
            local_max = fmax(local_max, data[i]);
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0;
        for (int i = 0; blockDim.x * i + threadIdx.x < kv_len; i++) {
            data[i] = __expf(data[i] - s_max);
            local_sum += data[i];
        }
        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);
        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < kv_len; i++) {
            qk_offset =
                ((blockIdx.y * head_num + blockIdx.z) * seq_len + seq_id) * kv_len + blockDim.x * i + threadIdx.x;
            qk_buf_[qk_offset] = (blockDim.x * i + threadIdx.x)<enc_len? (T)(data[i] * s_mean) : 0.f;
        }
    }
}
/*
template<typename T, int ITEMS_PER_THREAD>
__global__ void softmax_kernel_v4_half2(
    T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, const T scalar)
{
    using T2 = typename TypeConverter<T>::Type;
    T2* qk_buf_half2 = (T2*)qk_buf_;
    const T2* attr_mask_half2 = (const T2*)attr_mask;

    for (int seq_id = blockIdx.x; seq_id < seq_len; seq_id += gridDim.x) {
        T2 data[ITEMS_PER_THREAD];
        int qk_offset;
        __shared__ float s_mean, s_max;
        float local_max = -1e20f;
        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len / 2) && i < ITEMS_PER_THREAD; i++) {
            qk_offset = ((blockIdx.y * head_num + blockIdx.z) * seq_len + seq_id) * (seq_len / 2) + blockDim.x * i
                        + threadIdx.x;
            int mask_offset = (blockIdx.y * seq_len + seq_id) * (seq_len / 2) + blockDim.x * i + threadIdx.x;

            T2 qk = qk_buf_half2[qk_offset];
            T2 mask_val = ldg(&attr_mask_half2[mask_offset]);
            mask_val = hmul2<T2>(hsub2<T2>(float2type2<T2>(1.0f), mask_val), float2type2<T2>(-10000.0f));

            data[i] = hadd2<T2>(hmul2<T2>(qk, type2type2<T, T2>(scalar)), mask_val);

            local_max = fmax(local_max, fmax((float)data[i].x, (float)data[i].y));
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0;
        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len / 2) && i < ITEMS_PER_THREAD; i++) {
            data[i] = hexp2<T2>(hsub2<T2>(data[i], float2type2<T2>(s_max)));
            local_sum += (float)(data[i].x + data[i].y);
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);

        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len / 2) && i < ITEMS_PER_THREAD; i++) {
            qk_offset = ((blockIdx.y * head_num + blockIdx.z) * seq_len + seq_id) * (seq_len / 2) + blockDim.x * i
                        + threadIdx.x;
            qk_buf_half2[qk_offset] = hmul2<T2>(data[i], float2type2<T2>(s_mean));
        }
    }
}

template<typename T, int ITEMS_PER_THREAD, int NUM>
__global__ void softmax_kernel_v5_half2(
    T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, const T scalar)
{
    using T2 = typename TypeConverter<T>::Type;
    T2* qk_buf_half2 = (T2*)qk_buf_;
    const T2* attr_mask_half2 = (const T2*)attr_mask;

    for (int seq_id = blockIdx.x; seq_id < seq_len; seq_id += gridDim.x * NUM) {
        T2 data[NUM][ITEMS_PER_THREAD];

        int qk_offset[NUM];

        __shared__ float s_sum[NUM], s_max[NUM];
        float local_max[NUM];
#pragma unroll
        for (int j = 0; j < NUM; j++) {
            local_max[j] = -1e20f;
        }

        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len / 2) && i < ITEMS_PER_THREAD; i++) {
            int mask_offset[NUM];
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                qk_offset[j] = ((blockIdx.y * head_num + blockIdx.z) * seq_len + seq_id + j * gridDim.x) * (seq_len / 2)
                               + blockDim.x * i + threadIdx.x;
                mask_offset[j] =
                    (blockIdx.y * seq_len + seq_id + j * gridDim.x) * (seq_len / 2) + blockDim.x * i + threadIdx.x;
            }

            T2 mask_val[NUM];
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                mask_val[j] = ldg(&attr_mask_half2[mask_offset[j]]);
            }

            T2 qk[NUM];
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                qk[j] = qk_buf_half2[qk_offset[j]];
            }

#pragma unroll
            for (int j = 0; j < NUM; j++) {
                mask_val[j] = hmul2<T2>(hsub2<T2>(float2type2<T2>(1.0f), mask_val[j]), float2type2<T2>(-10000.0f));
            }

#pragma unroll
            for (int j = 0; j < NUM; j++) {
                data[j][i] = hadd2<T2>(hmul2<T2>(qk[j], type2type2<T, T2>(scalar)), mask_val[j]);
                local_max[j] = fmax(local_max[j], fmax((float)data[j][i].x, (float)data[j][i].y));
            }
        }

        if (blockDim.x <= 32) {
            warpReduceMaxV2<float, NUM>(local_max);
        }
        else {
            blockReduceMaxV2<float, NUM>(local_max);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                s_max[j] = local_max[j];
            }
        }
        __syncthreads();

        float local_sum[NUM];
#pragma unroll
        for (int j = 0; j < NUM; j++) {
            local_sum[j] = {0.f};
        }

        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len / 2) && i < ITEMS_PER_THREAD; i++) {
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                data[j][i] = hexp2<T2>(hsub2<T2>(data[j][i], float2type2<T2>(s_max[j])));
            }

#pragma unroll
            for (int j = 0; j < NUM; j++) {
                local_sum[j] += (float)(data[j][i].x + data[j][i].y);
            }
        }

        if (blockDim.x <= 32) {
            warpReduceSumV2<float, NUM>(local_sum);
        }
        else {
            blockReduceSumV2<float, NUM>(local_sum);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                s_sum[j] = __fdividef(1.0f, local_sum[j] + 1e-6f);
            }
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len / 2) && i < ITEMS_PER_THREAD; i++) {
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                qk_offset[j] = ((blockIdx.y * head_num + blockIdx.z) * seq_len + seq_id + j * gridDim.x) * (seq_len / 2)
                               + blockDim.x * i + threadIdx.x;
            }

#pragma unroll
            for (int j = 0; j < NUM; j++) {
                qk_buf_half2[qk_offset[j]] = hmul2<T2>(data[j][i], float2type2<T2>(s_sum[j]));
            }
        }
    }
}

*/
#define SOFTMAX_KERNEL(ITEMS_PER_THREAD)                                                                               \
    block.x /= ITEMS_PER_THREAD;                                                                                       \
    assert(block.x <= 1024);                                                                                           \
    if (is_half2) {                                                                                                    \
    }                                                                                                                  \
    else {                                                                                                             \
        softmax_kernel_v4<ITEMS_PER_THREAD, T, T_IN>                                                                   \
            <<<grid, block, 0, stream>>>(buffer, buffer_src, attr_mask, isCrossAtten, batch_size, head_num, seq_len, kv_len, scalar);        \
    }

template<typename T, typename T_IN>
void invokeMaskedSoftMax(T* buffer,
                         const T_IN* buffer_src,
                         const int* attr_mask,
                         const bool isCrossAtten,
                         const int batch_size,
                         const int seq_len,
                         const int kv_len,
                         const int head_num,
                         const T scalar,
                         hipStream_t stream)
{

    dim3 grid(seq_len, batch_size, head_num);
    if (batch_size * head_num > 360) {
        grid.x = ceil(float(seq_len) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && kv_len % 2 == 0;
    dim3 block((kv_len / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 3072 && block.x <= 4096) {
        SOFTMAX_KERNEL(4)
    }
    if (block.x > 2048) {
        SOFTMAX_KERNEL(3)
    }
    else if (block.x > 1024) {
        SOFTMAX_KERNEL(2)
    }
    else if (block.x > 0) {
        SOFTMAX_KERNEL(1)
    }
    else {
        // FT_CHECK(seq_len <= 4096);
    }
}

template void invokeMaskedSoftMax(float* buffer,
                         const float* buffer_src,
                         const int* attr_mask,
                         const bool isCrossAtten,
                         const int batch_size,
                         const int seq_len,
                         const int kv_len,
                         const int head_num,
                         const float scalar,
                         hipStream_t stream);

template<typename T>
__global__ void
transpose(T* src, T* dst, const int batch_size, const int seq_len, const int head_num, const int size_per_head)
{
    int batch_id = blockIdx.x / (head_num * seq_len);
    int seq_id = blockIdx.x % seq_len;
    int head_id = (blockIdx.x % (head_num * seq_len)) / seq_len;
    dst[batch_id * (head_num * seq_len * size_per_head) + seq_id * head_num * size_per_head + head_id * size_per_head
        + threadIdx.x] = src[blockIdx.x * size_per_head + threadIdx.x];
}

template<typename T>
void invokeTransposeQKV(T* dst,
                        T* src,
                        const int batch_size,
                        const int seq_len,
                        const int head_num,
                        const int size_per_head,
                        hipStream_t stream)
{
    dim3 grid, block;
    if (sizeof(T) == 2) {
        int seq_per_block = 1;
        grid.x = batch_size * head_num * seq_len / seq_per_block;
        while (seq_per_block < 4 && grid.x % 2 == 0) {
            grid.x /= 2;
            seq_per_block *= 2;
        }

        // FT_CHECK(grid.x * seq_per_block == batch_size * head_num * seq_len);

        if (seq_per_block * size_per_head % 2 == 0) {
            block.x = seq_per_block * size_per_head / 2;
            if (std::is_same<T, half>::value) {
                transpose<half2><<<grid, block, 0, stream>>>(
                    (half2*)src, (half2*)dst, batch_size, seq_len, head_num, size_per_head / 2);
            }
#ifdef ENABLE_BF16
            else {
                transpose<__hip_bfloat162><<<grid, block, 0, stream>>>(
                    (__hip_bfloat162*)src, (__hip_bfloat162*)dst, batch_size, seq_len, head_num, size_per_head / 2);
            }
#endif
        }
        else {
            block.x = seq_per_block * size_per_head;
            transpose<T><<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head);
        }
    }
    else {
        const int seq_per_block = 1;
        grid.x = batch_size * head_num * seq_len / seq_per_block;
        block.x = seq_per_block * size_per_head;
        transpose<T><<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head);
    }
}
template 
void invokeTransposeQKV(float* dst,
                        float* src,
                        const int batch_size,
                        const int seq_len,
                        const int head_num,
                        const int size_per_head,
                        hipStream_t stream);


template<typename T>
__global__ void addBiasResidual(T* output, const T* input, const T* bias, const int m, const int n)
{
    const int col_index = blockIdx.y * blockDim.x + threadIdx.x;
    if (col_index < n) {
        T bias_val = (bias == nullptr) ? (T)(0.0f) : bias[col_index];
        output[blockIdx.x * n + col_index] =
            output[blockIdx.x * n + col_index] + input[blockIdx.x * n + col_index] + bias_val;
    }
}

template<typename T>
void invokeAddBiasResidual(T* output, const T* input, const T* bias, const int m, const int n, hipStream_t stream)
{
    int blocks_per_row = ceil(float(n) / 1024);
    dim3 grid(m, blocks_per_row);
    dim3 block(min(n, 1024));
    addBiasResidual<<<grid, block, 0, stream>>>(output, input, bias, m, n);
}
template void invokeAddBiasResidual(float* output, const float* input, const float* bias, const int m, const int n, hipStream_t stream);

template<typename T>
__global__ void generalLayerNorm(
    const T* __restrict input, const T* __restrict gamma, const T* __restrict beta, T* output, int m, int n)
{
    const int tid = threadIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;

    float local_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        local_sum += (float)(ldg(&input[blockIdx.x * n + i]));
    }

    mean = blockReduceSum(local_sum);

    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    float local_var_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        float diff = (float)(ldg(&input[blockIdx.x * n + i])) - s_mean;
        local_var_sum += diff * diff;
    }
    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / n + 1e-5f);
    }
    __syncthreads();

    for (int i = tid; i < n; i += blockDim.x) {
        float beta_val = (beta == nullptr) ? 0.0f : (float)ldg(&beta[i]);
        output[blockIdx.x * n + i] =
            (T)((((float)input[blockIdx.x * n + i] - s_mean) * s_variance) * (float)(ldg(&gamma[i])) + beta_val);
    }
}

template<typename T>
void invokeGeneralLayerNorm(T* out,
                            const T* input,
                            const T* gamma,
                            const T* beta,
                            const int m,
                            const int n,
                            hipStream_t stream,
                            int opt_version)
{
    dim3 grid(m);
    if (false) {
        
    }
    else {
        dim3 block(min(n, 1024));

        /* For general cases, n is equal to hidden_units, e.g., 512/1024.
            Since we have warp shuffle inside the code, block.x % 32 should be 0.
        */
        if (n % 32 != 0) {
            block.x = 1024;
        }

        /* should pay attention to the rsqrt precision*/
        generalLayerNorm<T><<<grid, block, 0, stream>>>(input, gamma, beta, out, m, n);  // For gpt-3
    }
}

#undef HALF_LAYERNORM_OPT
#undef HALF_LAYERNORM_OPT2

template void invokeGeneralLayerNorm(float* out,
                                     const float* input,
                                     const float* gamma,
                                     const float* beta,
                                     const int m,
                                     const int n,
                                     hipStream_t stream,
                                     int opt_version);
template void invokeGeneralLayerNorm(half* out,
                                     const half* input,
                                     const half* gamma,
                                     const half* beta,
                                     const int m,
                                     const int n,
                                     hipStream_t stream,
                                     int opt_version);
#ifdef ENABLE_BF16
template void invokeGeneralLayerNorm(__hip_bfloat16* out,
                                     const __hip_bfloat16* input,
                                     const __hip_bfloat16* gamma,
                                     const __hip_bfloat16* beta,
                                     const int m,
                                     const int n,
                                     hipStream_t stream,
                                     int opt_version);
#endif
